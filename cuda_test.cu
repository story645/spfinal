#include "cuda_proximity.h"
#include "cuda_timer.h"
#include <cudpp.h>
#include <cutil.h>
#include "reduce_kernel.h"
#include <time.h>
#include "cuda_test.h"


void testall(float *data, unsigned int nSamples, unsigned int dim, 
             unsigned int dmin, unsigned int dmax, 
	     unsigned int BS, unsigned int LSH, unsigned int RESULT)
{
	unsigned int nQueries = nSamples*.20;
	unsigned int K = 250;
	
	printf("nSamples=%d, nQueries=%d, dims=%d, K=%d\n",
	nSamples, nQueries, dim, K);
	
	float* query = NULL;
	unsigned int* KNNResult = NULL;
	unsigned int* KNNResult_query = NULL;
	
	
	CPUMALLOC((void**)&query, sizeof(float) * nQueries * dim);
	CPUMALLOC((void**)&KNNResult, sizeof(unsigned int) * nSamples * K);
	CPUMALLOC((void**)&KNNResult_query, sizeof(unsigned int) * nQueries * K);
	
	
	
	
	//pick random samples for the queries
	uint init;
	uint qi = 0;
	for(uint q = 0; q < nQueries; q++){
	     init = (rand() % nSamples)*dim;
	     for(uint di=init; di<(init+dim); di++){
	     query[qi] = data[di];
	     qi++;
	     }
	}
	
	
	float* d_data = NULL;
	float* d_query = NULL;
	unsigned int* d_KNNResult = NULL;
	unsigned int* d_KNNResult_query = NULL;
	GPUMALLOC((void**)&d_data, sizeof(float) * nSamples * dim);
	GPUMALLOC((void**)&d_query, sizeof(float) * nQueries * dim);
	GPUMALLOC((void**)&d_KNNResult, sizeof(unsigned int) * nSamples * K);
	GPUMALLOC((void**)&d_KNNResult_query, sizeof(unsigned int) * nQueries * K);
	
	TOGPU(d_data, data, sizeof(float) * nSamples * dim);
	TOGPU(d_query, query, sizeof(float) * nQueries * dim);


    	if(BS){
	bruteforce(K, dim, nSamples, nQueries, d_data, d_query,
	           KNNResult, d_KNNResult, KNNResult_query,
		   d_KNNResult_query, RESULT);
	}

	if(LSH ){
	locsenhash(K, dim, nSamples, nQueries, d_data, d_query,
                   KNNResult, d_KNNResult, KNNResult_query,
                   d_KNNResult_query, RESULT);
	}
	
	GPUFREE(d_data);
	GPUFREE(d_KNNResult);
	CPUFREE(data);
	CPUFREE(KNNResult);
	
	GPUFREE(d_query);
	GPUFREE(d_KNNResult_query);
	CPUFREE(query);
	CPUFREE(KNNResult_query);
	
}


void bruteforce(unsigned int K, unsigned int dim, 
                unsigned int nSamples, unsigned int nQueries, 
		float* d_data, float* d_query, 
		unsigned int* KNNResult, unsigned int* d_KNNResult, 
		unsigned int* KNNResult_query, unsigned int* d_KNNResult_query, 
		unsigned int RESULT){
      //data points self query using radixsort
      unsigned int timer3 = 0;
      startTimer(&timer3);
      proximityComputation_bruteforce2(d_data, nSamples, d_data, nSamples,
      dim, K, 0.0f, d_KNNResult);
      FROMGPU(KNNResult, d_KNNResult, sizeof(unsigned int) * nSamples * K);
      endTimer("brute-force KNN - data points self query- using radixsort", &timer3);
      if(RESULT){printResults("knn_bf2.txt", KNNResult, nSamples, K);}
	
	
      //separate data/query points using radixsort	 
      unsigned int timer4 = 0;
      startTimer(&timer4);
      proximityComputation_bruteforce2(d_data, nSamples, d_query, nQueries,
      dim, K, 0.0f, d_KNNResult_query);
      FROMGPU(KNNResult_query, d_KNNResult_query, sizeof(unsigned int) * nQueries * K);
      endTimer("brute-force KNN - separate data/query points - using radixsort", &timer4);
      if(RESULT){printResults("knn_query_bf2.txt", KNNResult_query, nQueries, K);}
}

void locsenhash(unsigned int K, unsigned int dim,
            unsigned int nSamples, unsigned int nQueries,
	    float* d_data, float* d_query,
	    unsigned int* KNNResult, unsigned int* d_KNNResult,
            unsigned int* KNNResult_query, unsigned int* d_KNNResult_query, 
	    unsigned int RESULT){
	    
	    float* h_lower = NULL;
	    float* h_upper = NULL;
	    CPUMALLOC((void**)&h_lower, sizeof(float) * dim);
	    CPUMALLOC((void**)&h_upper, sizeof(float) * dim);
	    for(unsigned int i = 0; i < dim; ++i){
	    		 h_upper[i] = 1;
	    		 h_lower[i] = 0;
	    }
	    int LSH_L = 5;
	    
	    //data points self query
	    unsigned int timer1 = 0;
	    startTimer(&timer1);
	    proximityComputation_LSH(d_data, nSamples, d_data,
	    				     nSamples, dim, K, LSH_L, 0.0f, h_upper, h_lower, d_KNNResult);
	    FROMGPU(KNNResult, d_KNNResult, sizeof(unsigned int) * nSamples * K);
	    endTimer("LSH KNN -data point self query", &timer1);
	    if(RESULT){printResults("knn_lsh.txt", KNNResult, nSamples, K);}

	    unsigned int timer2 = 0;
	    startTimer(&timer2);
	    proximityComputation_LSH(d_data, nSamples, d_query, nQueries, dim, K,
	                              LSH_L, 0.0f, h_upper, h_lower, d_KNNResult_query);
	    FROMGPU(KNNResult_query, d_KNNResult_query, sizeof(unsigned int) * nQueries * K);
	    endTimer("LSH KNN - separte data/query points", &timer2);
	    if(RESULT){printResults("knn_query_lsh.txt", KNNResult_query, nQueries, K);}

	   CPUFREE(h_lower);
	   CPUFREE(h_upper); 
}
	    
void printResults(char *filename, unsigned int* output, 
                  unsigned int nLines, unsigned int K){
     FILE * fileout = fopen(filename, "w");
     for(uint i =0; i<nLines; ++i){
     	      for(uint j=0; j < K; ++j){
     	      	       fprintf(fileout, "%d", output[j * nLines + i]);
	      }	       
     fprintf(fileout, "\n");
     }
     fclose(fileout);
}

