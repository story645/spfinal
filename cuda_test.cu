#include "cuda_proximity.h"
#include "cuda_timer.h"
#include <cudpp.h>
#include <cutil.h>
#include "reduce_kernel.h"
#include <time.h>
#include "cuda_test.h"


void testall(float *data, uint nSamples, uint dim, uint dmin, uint
dmax, uint BS, uint LSH, uint RESULT)
{
	unsigned int nQueries = nSamples*.20;
	unsigned int K = 250;
	
	printf("nSamples=%d, nQueries=%d, dims=%d, K=%d\n",
	nSamples, nQueries, dim, K);
	
	float* query = NULL;
	unsigned int* KNNResult = NULL;
	unsigned int* KNNResult_query = NULL;
	
	
	CPUMALLOC((void**)&query, sizeof(float) * nQueries * dim);
	CPUMALLOC((void**)&KNNResult, sizeof(unsigned int) * nSamples * K);
	CPUMALLOC((void**)&KNNResult_query, sizeof(unsigned int) * nQueries * K);
	
	
	
	
	//pick random samples for the queries
	uint init;
	uint qi = 0;
	for(uint q = 0; q < nQueries; q++){
	     init = (rand() % nSamples)*dim;
	     for(uint di=init; di<(init+dim); di++){
	     query[qi] = data[di];
	     qi++;
	     }
	}
	
	
	float* d_data = NULL;
	float* d_query = NULL;
	unsigned int* d_KNNResult = NULL;
	unsigned int* d_KNNResult_query = NULL;
	GPUMALLOC((void**)&d_data, sizeof(float) * nSamples * dim);
	GPUMALLOC((void**)&d_query, sizeof(float) * nQueries * dim);
	GPUMALLOC((void**)&d_KNNResult, sizeof(unsigned int) * nSamples * K);
	GPUMALLOC((void**)&d_KNNResult_query, sizeof(unsigned int) * nQueries * K);
	
	TOGPU(d_data, data, sizeof(float) * nSamples * dim);
	TOGPU(d_query, query, sizeof(float) * nQueries * dim);


    	if(BS)
	{
		//data points self query using radixsort

		unsigned int timer3 = 0;
		startTimer(&timer3);
		
		proximityComputation_bruteforce2(d_data, nSamples, d_data, nSamples, dim, K, 0.0f, d_KNNResult);	
		FROMGPU(KNNResult, d_KNNResult, sizeof(unsigned int) * nSamples * K);
		endTimer("brute-force KNN -  data points self query- using radixsort", &timer3);	

		if(RESULT)
		{
			FILE* file3 = fopen("knn_bf2.txt", "w");
			for(unsigned int i = 0; i < nSamples; ++i)
			{
				for(unsigned int j = 0; j < K; ++j)
				{
					fprintf(file3, "%d ", KNNResult[j * nSamples + i]);
				}
				fprintf(file3, "\n");
			}
			fclose(file3);
		}
		
		//separate data/query points using radixsort		
		unsigned int timer4 = 0;
		startTimer(&timer4);
		
		proximityComputation_bruteforce2(d_data, nSamples, d_query, nQueries, dim, K, 0.0f, d_KNNResult_query);
		FROMGPU(KNNResult_query, d_KNNResult_query, sizeof(unsigned int) * nQueries * K);
		endTimer("brute-force KNN - separate data/query points - using radixsort", &timer4);
		
		if(RESULT)
		{
			FILE* file4 = fopen("knn_query_bf2.txt", "w");
			for(unsigned int i = 0; i < nQueries; ++i)
			{
				for(unsigned int j = 0; j < K; ++j)
				{
					fprintf(file4, "%d ", KNNResult_query[j * nQueries + i]);
				}
				fprintf(file4, "\n");
			}
			fclose(file4);
		}		
	}

	if(LSH )
	{
		float* h_lower = NULL;
		float* h_upper = NULL;
		CPUMALLOC((void**)&h_lower, sizeof(float) * dim);
		CPUMALLOC((void**)&h_upper, sizeof(float) * dim);
		
		for(unsigned int i = 0; i < dim; ++i)
		{
			h_upper[i] = 1;
			h_lower[i] = 0;
		}
		
		int LSH_L = 5;

		//data points self query
		unsigned int timer1 = 0;
		startTimer(&timer1);
		
		proximityComputation_LSH(d_data, nSamples, d_data, nSamples, dim, K, LSH_L, 0.0f, h_upper, h_lower, d_KNNResult);
		FROMGPU(KNNResult, d_KNNResult, sizeof(unsigned int) * nSamples * K);
		endTimer("LSH KNN -data point self query", &timer1);
		
		if(RESULT)
		{
			FILE* file1 = fopen("knn_lsh.txt", "w");
			for(unsigned int i = 0; i < nSamples; ++i)
			{
				for(unsigned int j = 0; j < K; ++j)
				{
					fprintf(file1, "%d ", KNNResult[j * nSamples + i]);
				}
				fprintf(file1, "\n");
			}
			fclose(file1);
		}
	
	
		unsigned int timer2 = 0;
		startTimer(&timer2);
		proximityComputation_LSH(d_data, nSamples, d_query, nQueries, dim, K, LSH_L, 0.0f, h_upper, h_lower, d_KNNResult_query);
		FROMGPU(KNNResult_query, d_KNNResult_query, sizeof(unsigned int) * nQueries * K);
		endTimer("LSH KNN - separte data/query points", &timer2);

		if(RESULT)
		{
			FILE* file2 = fopen("knn_query_lsh.txt", "w");
			for(unsigned int i = 0; i < nQueries; ++i)
			{
				for(unsigned int j = 0; j < K; ++j)
				{
					fprintf(file2, "%d ", KNNResult_query[j * nQueries + i]);
				}
				fprintf(file2, "\n");
			}
			fclose(file2);
		}		
		CPUFREE(h_lower);
		CPUFREE(h_upper);		
	}
	
	GPUFREE(d_data);
	GPUFREE(d_KNNResult);
	CPUFREE(data);
	CPUFREE(KNNResult);
	
	GPUFREE(d_query);
	GPUFREE(d_KNNResult_query);
	CPUFREE(query);
	CPUFREE(KNNResult_query);
	
}

